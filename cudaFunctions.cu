#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"
#include <stdio.h>


// __global__ void computeHistogramCUDA(int* data, int dataSize, int* histogram) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     // int chunck = (dataSize / (NUM_BLOCKS * THREADS_PER_BLOCK));
//     int stride = gridDim.x * blockDim.x;

//     for (int i = tid; i < dataSize; i += stride) {
//         atomicAdd(&histogram[data[i]], 1);
//     }
// }

__global__ void computeHistogramCUDA(int* data, int startIndex, int endIndex, int* histogram) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = tid + startIndex; i < endIndex; i += stride) {
        atomicAdd(&histogram[data[i]], 1);
    }
}

__global__  void initHist(int* h) {

  int index = threadIdx.x;
  h[index] = 0;

}

void computeOnGPU(int* data, int startIndex, int endIndex, int localSize, int** histogram) {

    printf("\nCCUUDDAA start:%d, NEW START:%d, end:%d NEW END:%d\n", startIndex, (endIndex - startIndex) + (localSize % 2), endIndex, localSize);
    startIndex = endIndex - startIndex + (localSize % 2);
    endIndex = localSize;

     // Allocate device memory on GPU for CUDA data and histogram from Host (CPU)
    hipError_t cudaStatus;
    int* cudaData = NULL;
    int* cudaHistogram = NULL;

    cudaStatus = hipMalloc((void**)&cudaData, localSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed for cudaData: %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMalloc((void**)&cudaHistogram, NUM_BINS * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed for cudaHistogram: %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to device (GPU memory)
    cudaStatus = hipMemcpy(cudaData, data, localSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA memcpy failed from host to device: %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // Initialize histogram on device memory 
    initHist<<<1, NUM_BINS>>>(cudaHistogram);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA memset failed for cudaHistogram: %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // Launch kernel for parallel histogram computation
    // computeHistogramCUDA<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(cudaData, localSize, cudaHistogram);
    computeHistogramCUDA<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(cudaData, startIndex, endIndex, cudaHistogram);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // // Synchronize to ensure all CUDA operations are completed
    // hipDeviceSynchronize();


    // Copy histogram result from device (GPU) to host (CPU)
    *histogram = (int*)malloc(NUM_BINS * sizeof(int));
    cudaStatus = hipMemcpy(*histogram, cudaHistogram, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA memcpy failed from device to host: %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // Clean up device memory
    if (hipFree(cudaData) != hipSuccess || hipFree(cudaHistogram) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

}